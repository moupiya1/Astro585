
#include <hip/hip_runtime.h>
extern "C"   // ensure function name to be left alone
{

    __global__ void normal_pdf_gpu(const double *x, double *y, unsigned int n)
    {
	// assumes a 2-d grid of 1-d blocks
	unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
        if(i<n)  y[i] = exp(-0.5*x[i]*x[i])*rsqrt(2.0*M_PI);
    }

}


